#include "SystemStructures.h"
#include "System.h"
#include "Plt_Arm_Node_Force.h"
#include "functor_plt_arm_node.h"
#include "functor_misc.h"

//tendril-like force
//The limit is plt_tndrl_intrct (small number)
//Force is applied to nodes
//We use the tndrl for imaging. 

void Plt_Arm_Node_Force(
	NodeInfoVecs& nodeInfoVecs,
	WLCInfoVecs& wlcInfoVecs,
	GeneralParams& generalParams,
	PltInfoVecs& pltInfoVecs,
	AuxVecs& auxVecs,
	RandVecs& randVecs) {


		thrust::fill(pltInfoVecs.nodeUnreducedForceX.begin(), pltInfoVecs.nodeUnreducedForceX.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeUnreducedForceY.begin(), pltInfoVecs.nodeUnreducedForceY.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeUnreducedForceZ.begin(), pltInfoVecs.nodeUnreducedForceZ.end(), 0.0);

		thrust::fill(pltInfoVecs.nodeReducedForceX.begin(), pltInfoVecs.nodeReducedForceX.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeReducedForceY.begin(), pltInfoVecs.nodeReducedForceY.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeReducedForceZ.begin(), pltInfoVecs.nodeReducedForceZ.end(), 0.0);

		//fill for image sort
    	thrust::fill(pltInfoVecs.nodeUnreducedId.begin(),pltInfoVecs.nodeUnreducedId.end(), generalParams.maxNodeCount);
		thrust::fill(pltInfoVecs.nodeImagingConnection.begin(),pltInfoVecs.nodeImagingConnection.end(), generalParams.maxNodeCount);
		thrust::counting_iterator<unsigned> counter(0);



		unsigned _seedplt = rand();

		thrust::counting_iterator<unsigned> index_sequence_plt_begin(_seedplt);
		thrust::transform(thrust::device, index_sequence_plt_begin, index_sequence_plt_begin + (generalParams.maxPltCount),
		randVecs.bucketPltStart.begin(), psrunifgen(0.0, 1.0));
        //Call the plt force on nodes functor
		//WARNING:
		//writes to unreduced vector entries from 0 to maxPltCount*plt_tndrl_intrct
        thrust::transform(
        	thrust::make_zip_iterator(
        		thrust::make_tuple(
					counter,
					auxVecs.idPlt_bucket.begin(),
					randVecs.bucketPltStart.begin(),
        			pltInfoVecs.pltLocX.begin(),
        			pltInfoVecs.pltLocY.begin(),
        			pltInfoVecs.pltLocZ.begin(),
					pltInfoVecs.pltForceX.begin(),
					pltInfoVecs.pltForceY.begin(),
					pltInfoVecs.pltForceZ.begin())),
        	thrust::make_zip_iterator(
        		thrust::make_tuple(
					counter,
					auxVecs.idPlt_bucket.begin(),
					randVecs.bucketPltStart.begin(),
        		 	pltInfoVecs.pltLocX.begin(),
        		 	pltInfoVecs.pltLocY.begin(),
        		 	pltInfoVecs.pltLocZ.begin(),
					pltInfoVecs.pltForceX.begin(),
					pltInfoVecs.pltForceY.begin(),
					pltInfoVecs.pltForceZ.begin())) + generalParams.maxPltCount,
         thrust::make_zip_iterator(
        	 thrust::make_tuple(
				 //DOES NOT RESET FORCES 
        		 pltInfoVecs.pltForceX.begin(),
        		 pltInfoVecs.pltForceY.begin(),
        		 pltInfoVecs.pltForceZ.begin())), 
             functor_plt_arm_node(
				generalParams.use_dynamic_plt_force,
				generalParams.CLM,
				generalParams.max_dynamic_plt_force,

                generalParams.plt_tndrl_intrct,
                generalParams.pltRForce,
                generalParams.pltForce,
                generalParams.pltR,

                generalParams.maxPltCount,
                generalParams.fiberDiameter,
		        generalParams.maxNodeCount,
                generalParams.maxIdCountFlag,
                generalParams.maxNeighborCount,
				generalParams.pltrelease,
				generalParams.plthandhand,

                thrust::raw_pointer_cast(nodeInfoVecs.nodeLocX.data()),
                thrust::raw_pointer_cast(nodeInfoVecs.nodeLocY.data()),
                thrust::raw_pointer_cast(nodeInfoVecs.nodeLocZ.data()),
                thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedForceX.data()),
                thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedForceY.data()),
                thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedForceZ.data()),

                thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedId.data()),
                thrust::raw_pointer_cast(pltInfoVecs.pltImagingConnection.data()),

                thrust::raw_pointer_cast(auxVecs.id_value_expanded_plt_intc.data()),//network neighbors
                thrust::raw_pointer_cast(auxVecs.keyBegin_plt_intc.data()),
                thrust::raw_pointer_cast(auxVecs.keyEnd_plt_intc.data()),

                thrust::raw_pointer_cast(pltInfoVecs.tndrlNodeId.data()),
                thrust::raw_pointer_cast(pltInfoVecs.tndrlNodeType.data()),
                thrust::raw_pointer_cast(nodeInfoVecs.isNodeInPltVol.data()),
                thrust::raw_pointer_cast(wlcInfoVecs.globalNeighbors.data()),
                thrust::raw_pointer_cast(wlcInfoVecs.lengthZero.data()),
                thrust::raw_pointer_cast(wlcInfoVecs.numOriginalNeighborsNodeVector.data()),

                thrust::raw_pointer_cast(pltInfoVecs.pltLocX.data()),
                thrust::raw_pointer_cast(pltInfoVecs.pltLocY.data()),
                thrust::raw_pointer_cast(pltInfoVecs.pltLocZ.data())) ); 

		
        //now call a sort by key followed by a reduce by key to figure out which nodes are have force applied.
        //then make a functor that takes the id and force (4 tuple) and takes that force and adds it to the id'th entry in nodeInfoVecs.nodeForceX,Y,Z

		
		unsigned total_num_arms = pltInfoVecs.nodeImagingConnection.size();
		
		//correspondance kept between nodeUnreducedId and pltImagingConnection
		thrust::stable_sort_by_key(pltInfoVecs.nodeUnreducedId.begin(), pltInfoVecs.nodeUnreducedId.end(),
        			thrust::make_zip_iterator(
        				thrust::make_tuple(
							pltInfoVecs.pltImagingConnection.begin(),
        					pltInfoVecs.nodeUnreducedForceX.begin(),
        					pltInfoVecs.nodeUnreducedForceY.begin(),
							pltInfoVecs.nodeUnreducedForceZ.begin())), thrust::less<unsigned>());
							
		//now nodeImagingConnection contains the corresponding nodes to pltImagingConnection
    	thrust::copy(pltInfoVecs.nodeUnreducedId.begin(),pltInfoVecs.nodeUnreducedId.begin() + total_num_arms, pltInfoVecs.nodeImagingConnection.begin());

    	pltInfoVecs.numConnections = thrust::count_if(
    	    pltInfoVecs.nodeImagingConnection.begin(),
    	    pltInfoVecs.nodeImagingConnection.end(), is_less_than(generalParams.maxNodeCount) );


		//reduce and apply force
 		unsigned endKey = thrust::get<0>(
 			thrust::reduce_by_key(
 				pltInfoVecs.nodeUnreducedId.begin(),
 				pltInfoVecs.nodeUnreducedId.begin() + total_num_arms,
 			thrust::make_zip_iterator(
 				thrust::make_tuple(
 					pltInfoVecs.nodeUnreducedForceX.begin(),
 					pltInfoVecs.nodeUnreducedForceY.begin(),
 					pltInfoVecs.nodeUnreducedForceZ.begin())),
 			pltInfoVecs.nodeReducedId.begin(),
 			thrust::make_zip_iterator(
 				thrust::make_tuple(//need t check
 					pltInfoVecs.nodeReducedForceX.begin(),
 					pltInfoVecs.nodeReducedForceY.begin(),
 					pltInfoVecs.nodeReducedForceZ.begin())),
 			thrust::equal_to<unsigned>(), CVec3Add())) - pltInfoVecs.nodeReducedId.begin();//binary_pred, binary_op

		//std::cout<<"endkey: "<< endKey<<std::endl;
        thrust::for_each(
        	thrust::make_zip_iterator(//1st begin
        		thrust::make_tuple(
        			pltInfoVecs.nodeReducedId.begin(),
        			pltInfoVecs.nodeReducedForceX.begin(),
        			pltInfoVecs.nodeReducedForceY.begin(),
        			pltInfoVecs.nodeReducedForceZ.begin())),
        	thrust::make_zip_iterator(//1st end
        		thrust::make_tuple(
        			pltInfoVecs.nodeReducedId.begin(),
        			pltInfoVecs.nodeReducedForceX.begin(),
        			pltInfoVecs.nodeReducedForceY.begin(),
        			pltInfoVecs.nodeReducedForceZ.begin())) + endKey,
			functor_add_UCVec3_CVec3(
        		thrust::raw_pointer_cast(nodeInfoVecs.nodeForceX.data()),
        		thrust::raw_pointer_cast(nodeInfoVecs.nodeForceY.data()),
        		thrust::raw_pointer_cast(nodeInfoVecs.nodeForceZ.data())));
				
};