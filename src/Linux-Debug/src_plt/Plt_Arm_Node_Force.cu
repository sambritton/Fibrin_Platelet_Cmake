#include "hip/hip_runtime.h"
#include "SystemStructures.h"
#include "System.h"
#include "Plt_Arm_Node_Force.h"
#include "functor_plt_arm_node.h"
#include "functor_misc.h"

//tendril-like force
//The limit is plt_tndrl_intrct (small number)
//Force is applied to nodes
//We use the tndrl for imaging. 

void Plt_Arm_Node_Force(
	NodeInfoVecs& nodeInfoVecs,
	WLCInfoVecs& wlcInfoVecs,
	GeneralParams& generalParams,
	PltInfoVecs& pltInfoVecs,
	AuxVecs& auxVecs,
	RandVecs& randVecs) {


		thrust::fill(pltInfoVecs.nodeUnreducedForceX.begin(), pltInfoVecs.nodeUnreducedForceX.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeUnreducedForceY.begin(), pltInfoVecs.nodeUnreducedForceY.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeUnreducedForceZ.begin(), pltInfoVecs.nodeUnreducedForceZ.end(), 0.0);

		thrust::fill(pltInfoVecs.nodeReducedForceX.begin(), pltInfoVecs.nodeReducedForceX.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeReducedForceY.begin(), pltInfoVecs.nodeReducedForceY.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeReducedForceZ.begin(), pltInfoVecs.nodeReducedForceZ.end(), 0.0);

		//fill for image sort
    	thrust::fill(pltInfoVecs.nodeUnreducedId.begin(),pltInfoVecs.nodeUnreducedId.end(), generalParams.maxNodeCount);
		thrust::fill(pltInfoVecs.nodeImagingConnection.begin(),pltInfoVecs.nodeImagingConnection.end(), generalParams.maxNodeCount);
		thrust::counting_iterator<unsigned> counter(0);



		unsigned _seedplt = rand();

		thrust::counting_iterator<unsigned> index_sequence_plt_begin(_seedplt);
		thrust::transform(thrust::device, index_sequence_plt_begin, index_sequence_plt_begin + (generalParams.maxPltCount),
		randVecs.bucketPltStart.begin(), psrunifgen(0.0, 1.0));
        //Call the plt force on nodes functor
		//WARNING:
		//writes to unreduced vector entries from 0 to maxPltCount*plt_tndrl_intrct
        thrust::transform(
        	thrust::make_zip_iterator(
        		thrust::make_tuple(
					counter,
					auxVecs.idPlt_bucket.begin(),
					randVecs.bucketPltStart.begin(),
        			pltInfoVecs.pltLocX.begin(),
        			pltInfoVecs.pltLocY.begin(),
        			pltInfoVecs.pltLocZ.begin(),
					pltInfoVecs.pltForceX.begin(),
					pltInfoVecs.pltForceY.begin(),
					pltInfoVecs.pltForceZ.begin())),
        	thrust::make_zip_iterator(
        		thrust::make_tuple(
					counter,
					auxVecs.idPlt_bucket.begin(),
					randVecs.bucketPltStart.begin(),
        		 	pltInfoVecs.pltLocX.begin(),
        		 	pltInfoVecs.pltLocY.begin(),
        		 	pltInfoVecs.pltLocZ.begin(),
					pltInfoVecs.pltForceX.begin(),
					pltInfoVecs.pltForceY.begin(),
					pltInfoVecs.pltForceZ.begin())) + generalParams.maxPltCount,
         thrust::make_zip_iterator(
        	 thrust::make_tuple(
				 //DOES NOT RESET FORCES 
        		 pltInfoVecs.pltForceX.begin(),
        		 pltInfoVecs.pltForceY.begin(),
        		 pltInfoVecs.pltForceZ.begin())), 
             functor_plt_arm_node(
				generalParams.use_dynamic_plt_force,
				generalParams.CLM,
				generalParams.max_dynamic_plt_force,

                generalParams.plt_tndrl_intrct,
                generalParams.pltRForce,
                generalParams.pltForce,
                generalParams.pltR,

                generalParams.maxPltCount,
                generalParams.fiberDiameter,
		        generalParams.maxNodeCount,
                generalParams.maxIdCountFlag,
                generalParams.maxNeighborCount,
				generalParams.pltrelease,
				generalParams.plthandhand,

                thrust::raw_pointer_cast(nodeInfoVecs.nodeLocX.data()),
                thrust::raw_pointer_cast(nodeInfoVecs.nodeLocY.data()),
                thrust::raw_pointer_cast(nodeInfoVecs.nodeLocZ.data()),
                thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedForceX.data()),
                thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedForceY.data()),
                thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedForceZ.data()),

                thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedId.data()),
                thrust::raw_pointer_cast(pltInfoVecs.pltImagingConnection.data()),

                thrust::raw_pointer_cast(auxVecs.id_value_expanded_plt_intc.data()),//network neighbors
                thrust::raw_pointer_cast(auxVecs.keyBegin_plt_intc.data()),
                thrust::raw_pointer_cast(auxVecs.keyEnd_plt_intc.data()),

                thrust::raw_pointer_cast(pltInfoVecs.tndrlNodeId.data()),
                thrust::raw_pointer_cast(pltInfoVecs.tndrlNodeType.data()),
                thrust::raw_pointer_cast(nodeInfoVecs.isNodeInPltVol.data()),
                thrust::raw_pointer_cast(wlcInfoVecs.globalNeighbors.data()),
                thrust::raw_pointer_cast(wlcInfoVecs.lengthZero.data()),
                thrust::raw_pointer_cast(wlcInfoVecs.numOriginalNeighborsNodeVector.data()),

                thrust::raw_pointer_cast(pltInfoVecs.pltLocX.data()),
                thrust::raw_pointer_cast(pltInfoVecs.pltLocY.data()),
                thrust::raw_pointer_cast(pltInfoVecs.pltLocZ.data())) ); 

		
        //now call a sort by key followed by a reduce by key to figure out which nodes are have force applied.
        //then make a functor that takes the id and force (4 tuple) and takes that force and adds it to the id'th entry in nodeInfoVecs.nodeForceX,Y,Z
		for (unsigned i = 0; i < pltInfoVecs.pltImagingConnection.size(); i++) {
			unsigned node_Id = pltInfoVecs.nodeUnreducedId[i];
			unsigned plt_id = pltInfoVecs.pltImagingConnection[i];
			if (node_Id < generalParams.maxNodeCount ){
				std::cout<<"pre_sort node: " << node_Id << " pltId: " << plt_id<< " iter: " <<i << std::endl;
			}
		}

		
		unsigned total_num_arms = pltInfoVecs.nodeImagingConnection.size();
		
		//correspondance kept between nodeUnreducedId and pltImagingConnection
		thrust::stable_sort_by_key(pltInfoVecs.nodeUnreducedId.begin(), pltInfoVecs.nodeUnreducedId.end(),
        			thrust::make_zip_iterator(
        				thrust::make_tuple(
							pltInfoVecs.pltImagingConnection.begin(),
        					pltInfoVecs.nodeUnreducedForceX.begin(),
        					pltInfoVecs.nodeUnreducedForceY.begin(),
							pltInfoVecs.nodeUnreducedForceZ.begin())), thrust::less<unsigned>());
							
		for (unsigned i = 0; i < pltInfoVecs.pltImagingConnection.size(); i++) {
			unsigned node_Id = pltInfoVecs.nodeUnreducedId[i];
			unsigned plt_id = pltInfoVecs.pltImagingConnection[i];
			if (node_Id < generalParams.maxNodeCount ){
				std::cout<<"post_sort node: " << node_Id << " pltId: " << plt_id<< " iter: " <<i << std::endl;
			}
		}
		//now nodeImagingConnection contains the corresponding nodes to pltImagingConnection
    	thrust::copy(pltInfoVecs.nodeUnreducedId.begin(),pltInfoVecs.nodeUnreducedId.begin() + total_num_arms, pltInfoVecs.nodeImagingConnection.begin());

    	pltInfoVecs.numConnections = thrust::count_if(
    	    pltInfoVecs.nodeImagingConnection.begin(),
    	    pltInfoVecs.nodeImagingConnection.end(), is_less_than(generalParams.maxNodeCount) );


		for (unsigned i = 0; i < pltInfoVecs.numConnections; i++) {
			unsigned node_Id = pltInfoVecs.nodeImagingConnection[i];
			unsigned plt_id = pltInfoVecs.pltImagingConnection[i];
			unsigned node_delete = pltInfoVecs.nodeUnreducedId[i];
			std::cout<<"node: " << node_Id << " pltId: " << plt_id<< " unused: " << node_delete << std::endl;

			//TEMP DELETE AFTER USE
			double locX = nodeInfoVecs.nodeLocX[node_Id];
			double locY = nodeInfoVecs.nodeLocY[node_Id];
			double locZ = nodeInfoVecs.nodeLocZ[node_Id];
			double pltX = pltInfoVecs.pltLocX[plt_id];
			double pltY = pltInfoVecs.pltLocY[plt_id];
			double pltZ = pltInfoVecs.pltLocZ[plt_id];
			double distX = locX - pltX;
			double distY = locY - pltY;
			double distZ = locZ - pltZ;
			
			double dist = sqrt(distX*distX + distY*distY + distZ * distZ);
			std::cout<<"dist true: "<< dist<<std::endl;
		} 
		/*for (unsigned i = 0; i < auxVecs.idPlt_bucket.size(); i++){
			std::cout<<"plt buckettndrl_1_2: "<<auxVecs.idPlt_bucket[i] << std::endl;
		}
		for (unsigned i = 0; i < pltInfoVecs.nodeImagingConnection.size(); i++){
			std::cout<<"plt nodeimaging_1_2: "<<pltInfoVecs.nodeImagingConnection[i] << std::endl;
		}
		for (unsigned i = 0; i < total_num_arms; i++){
			std::cout<<"plt nodeUnreducedId_1_2: "<<pltInfoVecs.nodeUnreducedId[i] << std::endl;
		}*/
//reduce and apply force
 		unsigned endKey = thrust::get<0>(
 			thrust::reduce_by_key(
 				pltInfoVecs.nodeUnreducedId.begin(),
 				pltInfoVecs.nodeUnreducedId.begin() + total_num_arms,
 			thrust::make_zip_iterator(
 				thrust::make_tuple(
 					pltInfoVecs.nodeUnreducedForceX.begin(),
 					pltInfoVecs.nodeUnreducedForceY.begin(),
 					pltInfoVecs.nodeUnreducedForceZ.begin())),
 			pltInfoVecs.nodeReducedId.begin(),
 			thrust::make_zip_iterator(
 				thrust::make_tuple(//need t check
 					pltInfoVecs.nodeReducedForceX.begin(),
 					pltInfoVecs.nodeReducedForceY.begin(),
 					pltInfoVecs.nodeReducedForceZ.begin())),
 			thrust::equal_to<unsigned>(), CVec3Add())) - pltInfoVecs.nodeReducedId.begin();//binary_pred, binary_op

		//std::cout<<"endkey: "<< endKey<<std::endl;
        thrust::for_each(
        	thrust::make_zip_iterator(//1st begin
        		thrust::make_tuple(
        			pltInfoVecs.nodeReducedId.begin(),
        			pltInfoVecs.nodeReducedForceX.begin(),
        			pltInfoVecs.nodeReducedForceY.begin(),
        			pltInfoVecs.nodeReducedForceZ.begin())),
        	thrust::make_zip_iterator(//1st end
        		thrust::make_tuple(
        			pltInfoVecs.nodeReducedId.begin(),
        			pltInfoVecs.nodeReducedForceX.begin(),
        			pltInfoVecs.nodeReducedForceY.begin(),
        			pltInfoVecs.nodeReducedForceZ.begin())) + endKey,
			functor_add_UCVec3_CVec3(
        		thrust::raw_pointer_cast(nodeInfoVecs.nodeForceX.data()),
        		thrust::raw_pointer_cast(nodeInfoVecs.nodeForceY.data()),
        		thrust::raw_pointer_cast(nodeInfoVecs.nodeForceZ.data())));
				
};