#include "SystemStructures.h"
#include "functor_advance_pos.h"
#include "System.h"
#include "Advance_Positions_Plt.h"


void Advance_Positions_Plt(
	PltInfoVecs& pltInfoVecs,
	GeneralParams& generalParams,
	RandVecs& randVecs) {

	unsigned _seedplt = rand();

	thrust::counting_iterator<unsigned> index_sequence_plt_begin(_seedplt);


	
 	thrust::transform(thrust::device, index_sequence_plt_begin, index_sequence_plt_begin + (generalParams.maxPltCount),
		randVecs.gaussianPltData.begin(), psrunifgen(-1.0, 1.0));

thrust::counting_iterator<unsigned> pltIndexBegin(0);

thrust::transform(
 thrust::make_zip_iterator(
	 thrust::make_tuple(
		 pltIndexBegin,
		 pltInfoVecs.pltLocX.begin(),
		 pltInfoVecs.pltLocY.begin(),
		 pltInfoVecs.pltLocZ.begin())),
 thrust::make_zip_iterator(
	 thrust::make_tuple(
		 pltIndexBegin,
		 pltInfoVecs.pltLocX.begin(),
		 pltInfoVecs.pltLocY.begin(),
		 pltInfoVecs.pltLocZ.begin())) + generalParams.maxPltCount,
 //second vector begin
 thrust::make_zip_iterator(
	 thrust::make_tuple(
		 randVecs.gaussianPltData.begin(),
		 pltInfoVecs.pltForceX.begin(),
		 pltInfoVecs.pltForceY.begin(),
		 pltInfoVecs.pltForceZ.begin())),
 //save result in third vector to test values
 thrust::make_zip_iterator(
	 thrust::make_tuple(
		 pltInfoVecs.pltLocX.begin(),
		 pltInfoVecs.pltLocY.begin(),
		 pltInfoVecs.pltLocZ.begin(),
		 pltInfoVecs.pltVelocity.begin())),
 functor_advance_pos(generalParams.dtTemp,
	 generalParams.viscousDamp_Plt,
	 generalParams.temperature,
	 generalParams.kB,
	 generalParams.pltMass,
	 generalParams.maxPltCount,
	 thrust::raw_pointer_cast(pltInfoVecs.isPltFixed.data())));


};
