#include "hip/hip_runtime.h"


#include "System.h"
#include "System_Builder.h"
#include "SystemStructures.h"
#include "Storage.h"


Storage::Storage(std::weak_ptr<System> a_system,
	std::weak_ptr<SystemBuilder> b_system ) {
	//std::cout << "FDM constructor" << std::endl;

	system = a_system;
	builder = b_system;

};



void Storage::save_params(void) {
	std::shared_ptr<System> sys = system.lock();
	if (sys) {

		//first create a new file using the current network strain
		
		std::string format = ".sta";
		
		std::string strain =  std::to_string(sys->generalParams.currentTime);
		std::string initial = "Params/Param_";
		std::ofstream ofs;
		std::string Filename = initial + strain + format;
		ofs.open(Filename.c_str());



		//unsigned maxNeighborCount = sys->generalParams.maxNeighborCount;
		unsigned maxNodeCount = sys->generalParams.maxNodeCount;
		unsigned originalNodeCount = sys->generalParams.originNodeCount;
		unsigned originalEdgeCount = sys->generalParams.originLinkCount;
		unsigned edgeCountDiscretize = sys->generalParams.originEdgeCount;
		//Now first place strain
		ofs << std::setprecision(5) <<std::fixed<< "time " << sys->generalParams.currentTime<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "minX " << sys->domainParams.minX<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "maxX " << sys->domainParams.maxX<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "minY " << sys->domainParams.minY<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "maxY " << sys->domainParams.maxY<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "minZ " << sys->domainParams.minX<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "maxZ " << sys->domainParams.maxX<<std::endl;
		
		
		ofs << std::setprecision(5) <<std::fixed<< "original_node_count " << originalNodeCount <<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "node_count_discretize " << maxNodeCount <<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "original_edge_count " << originalEdgeCount <<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "edge_count_discretize " << edgeCountDiscretize <<std::endl;
		
		//place nodes
		
		//thrust::copy(sys->nodeInfoVecs.nodeLocX.begin(),
		//	sys->nodeInfoVecs.nodeLocX.end(), hostLocX.begin());
		for (unsigned i = 0; i < sys->nodeInfoVecs.nodeLocX.size(); i++) {
			
			double x = sys->nodeInfoVecs.nodeLocX[i];
			double y = sys->nodeInfoVecs.nodeLocY[i];
			double z = sys->nodeInfoVecs.nodeLocZ[i];
			ofs << std::setprecision(5) <<std::fixed<< "node " << x << " " << y << " " << z <<std::endl;
		}
		
		//place plts
		for (unsigned i = 0; i < sys->pltInfoVecs.pltLocX.size(); i++) {
			double x = sys->pltInfoVecs.pltLocX[i];
			double y = sys->pltInfoVecs.pltLocY[i];
			double z = sys->pltInfoVecs.pltLocZ[i];
			ofs << std::setprecision(5) <<std::fixed<< "plt " << x << " " << y << " " << z <<std::endl;
		
		}
		//place force node is experiencing
		for (unsigned i = 0; i < sys->nodeInfoVecs.nodeLocX.size(); i++) {
			ofs << std::setprecision(5) <<std::fixed<< "force_on_node " << sys->nodeInfoVecs.sumForcesOnNode[i]<<std::endl;
		
		}

		//place original edges
		for (unsigned edge = 0; edge < sys->generalParams.originEdgeCount; edge++) {
			unsigned idL = sys->nodeInfoVecs.hostEdgeLeft[edge];
			unsigned idR = sys->nodeInfoVecs.hostEdgeRight[edge];
			ofs <<"original_edge_discretized " <<idL <<" "<< idR <<std::endl;
			
		}
				 
		//place added edges
		for (unsigned edge = sys->generalParams.originEdgeCount; edge < sys->generalParams.currentEdgeCount; edge++) {
			unsigned idL = sys->nodeInfoVecs.hostEdgeLeft[edge];
			unsigned idR = sys->nodeInfoVecs.hostEdgeRight[edge];
			ofs <<"added_edge " <<idL <<" "<< idR <<std::endl;
			
		}

		//original edge strain
		for (unsigned i = 0; i < sys->generalParams.originEdgeCount; i++ ){
			double val = sys->nodeInfoVecs.discretizedEdgeStrain[i];

			ofs << std::setprecision(5)<< std::fixed<<"original_edge_strain " << val <<std::endl;
		}
				
		//original edge alignment
		for (unsigned i = 0; i < sys->generalParams.originEdgeCount; i++ ){
			double val = sys->nodeInfoVecs.discretizedEdgeAlignment[i];
			ofs << std::setprecision(5)<< std::fixed<<"original_edge_alignment " << val <<std::endl;
		}

		//added edge strain
		for (unsigned i = sys->generalParams.originEdgeCount; i < sys->generalParams.currentEdgeCount; i++ ){
			double val = sys->nodeInfoVecs.discretizedEdgeStrain[i];
			ofs << std::setprecision(5)<< std::fixed<<"added_edge_strain " << val <<std::endl;
		}
		
		//added links per node.
		for (unsigned i = 0; i < sys->generalParams.maxNodeCount; i++ ){
			unsigned val = sys->wlcInfoVecs.currentNodeEdgeCountVector[i] - 
				sys->wlcInfoVecs.numOriginalNeighborsNodeVector[i];
			ofs << std::setprecision(5)<< std::fixed<<"bind_sites_per_node " << val <<std::endl;
		}



	}
};


void Storage::print_VTK_File() {

	std::shared_ptr<System> sys = system.lock();
	if (sys) {
		iteration+=1;
		unsigned digits = ceil(log10(iteration + 1));
		std::string format = ".vtk";
		std::string Number;
		std::string initial = "AnimationTest/FibrinNetwork_";
		std::ofstream ofs;
		if (digits == 1 || digits == 0) {
			Number = "0000" + std::to_string(iteration);
		}
		else if (digits == 2) {
			Number = "000" + std::to_string(iteration);
		}
		else if (digits == 3) {
			Number = "00" + std::to_string(iteration);
		}
		else if (digits == 4) {
			Number = "0" + std::to_string(iteration);
		}

		std::string Filename = initial + Number + format;

		ofs.open(Filename.c_str());


		unsigned maxNodeCount = sys->generalParams.maxNodeCount;
		unsigned maxNeighborCount = (sys->generalParams).maxNeighborCount;

		unsigned numEdges = sys->generalParams.currentEdgeCount;

		ofs << "# vtk DataFile Version 3.0" << std::endl;
		ofs << "Point representing Sub_cellular elem model" << std::endl;
		ofs << "ASCII" << std::endl << std::endl;
		ofs << "DATASET UNSTRUCTURED_GRID" << std::endl;


		ofs << "POINTS " << maxNodeCount << " float" << std::endl;
		for (unsigned i = 0; i< maxNodeCount; i++) { 
			double xPos = sys->nodeInfoVecs.nodeLocX[i];
			double yPos = sys->nodeInfoVecs.nodeLocY[i];
			double zPos = sys->nodeInfoVecs.nodeLocZ[i];

			ofs << std::setprecision(5) <<std::fixed<< xPos << " " << yPos << " " << zPos << " " << '\n'<< std::fixed;
		}
		//now plot particles


		unsigned numCells = numEdges;
		unsigned numNumsInCells = 3 * numEdges;


		ofs << "CELLS " << numCells << " " << numNumsInCells << std::endl;

		for (unsigned edge = 0; edge < numEdges; edge++) {
			unsigned idL = sys->nodeInfoVecs.hostEdgeLeft[edge];
			unsigned idR = sys->nodeInfoVecs.hostEdgeRight[edge];

			ofs<< 2 << " " << idL << " " << idR << std::endl;
		}
	/*	for (unsigned idA = 0; idA < maxNodeCount; idA++ ){

			unsigned beginIndex = idA * maxNeighborCount;
			unsigned endIndex = beginIndex + maxNeighborCount;
			for (unsigned i = beginIndex; i < endIndex; i++) {//currentSpringCount is the length of index and value vectors
				unsigned idB = sys->wlcInfoVecs.globalNeighbors[i];//look through possible neighbors. May contain ULONG_MAX

				//counts only half
				if ((idA < idB) && (idB < maxNodeCount) ) {
					ofs<< 2 << " " << idA << " " << idB << std::endl;
				}
			}
		}*/

		ofs << "CELL_TYPES " << numCells << std::endl;
		for (unsigned i = 0; i< numEdges; i++) {
			ofs << 3 << std::endl; //edge joining two points
		}



		//
		ofs << "CELL_DATA " << numCells << std::endl;
		ofs << "SCALARS Fiber_Strain double " << std::endl;
		ofs << "LOOKUP_TABLE default "  << std::endl;
		for (unsigned edge = 0; edge < numEdges; edge++) {
			unsigned idA = sys->nodeInfoVecs.hostEdgeLeft[edge];
			unsigned idB = sys->nodeInfoVecs.hostEdgeRight[edge];

			unsigned begin = idA * sys->generalParams.maxNeighborCount;
			unsigned end = begin + sys->generalParams.maxNeighborCount;
			double L0;
			for (unsigned i = begin; i < end; i++) {
				unsigned idTemp = sys->wlcInfoVecs.globalNeighbors[i];
				if (idTemp == idB){
					L0 = sys->wlcInfoVecs.lengthZero[i];
				}
			}
			double xL = sys->nodeInfoVecs.nodeLocX[idA];
			double yL = sys->nodeInfoVecs.nodeLocY[idA];
			double zL = sys->nodeInfoVecs.nodeLocZ[idA];
			double xR = sys->nodeInfoVecs.nodeLocX[idB];
			double yR = sys->nodeInfoVecs.nodeLocY[idB];
			double zR = sys->nodeInfoVecs.nodeLocZ[idB];

			double L1 = std::sqrt( (xL - xR)*(xL - xR)+(yL - yR)*(yL - yR)+(zL - zR)*(zL - zR));
			double strain = (L1 - L0) / L0;
			ofs << std::fixed << strain   << std::endl;

		}
	/*	for (unsigned idA = 0; idA < maxNodeCount; idA++ ){

			unsigned beginIndex = idA * maxNeighborCount;
			unsigned endIndex = beginIndex + maxNeighborCount;
			for (unsigned i = beginIndex; i < endIndex; i++) {//currentSpringCount is the length of index and value vectors
				unsigned idB = sys->wlcInfoVecs.globalNeighbors[i];//look through possible neighbors. May contain ULONG_MAX

				if ((idA < idB) && (idB < maxNodeCount) ) {
					__attribute__ ((unused)) unsigned index = idA * maxNeighborCount + idB;
					double L0 = sys->wlcInfoVecs.lengthZero[i];
					double xL = sys->nodeInfoVecs.nodeLocX[idA];
					double yL = sys->nodeInfoVecs.nodeLocY[idA];
					double zL = sys->nodeInfoVecs.nodeLocZ[idA];
					double xR = sys->nodeInfoVecs.nodeLocX[idB];
					double yR = sys->nodeInfoVecs.nodeLocY[idB];
					double zR = sys->nodeInfoVecs.nodeLocZ[idB];



					double L1 = std::sqrt( (xL - xR)*(xL - xR)+(yL - yR)*(yL - yR)+(zL - zR)*(zL - zR));
					double strain = (L1 - L0) / L0;
					ofs << std::fixed << strain   << std::endl;
				}
			}
		}*/

		ofs.close();

	}

	//now print platelets 
	if ((sys)) {
		unsigned digits = ceil(log10(iteration + 1));
		std::string format = ".vtk";
		std::string Number;
		std::string initial = "AnimationTest/Platelet";
		std::ofstream ofs;
		if (digits == 1 || digits == 0) {
			Number = "0000" + std::to_string(iteration);
		}
		else if (digits == 2) {
			Number = "000" + std::to_string(iteration);
		}
		else if (digits == 3) {
			Number = "00" + std::to_string(iteration);
		}
		else if (digits == 4) {
			Number = "0" + std::to_string(iteration);
		}

		std::string Filename = initial + Number + format;

		ofs.open(Filename.c_str());
		
	
		unsigned maxPltCount = sys->generalParams.maxPltCount;
		
		double xPos;
		double yPos;
		double zPos;
		
		ofs << "# vtk DataFile Version 3.0" << std::endl;
		ofs << "Point representing Sub_cellular elem model" << std::endl;
		ofs << "ASCII" << std::endl << std::endl;
		ofs << "DATASET UNSTRUCTURED_GRID" << std::endl;
		
		 
		ofs << "POINTS " << maxPltCount  << " float" << std::endl;
		for (unsigned i = 0; i< maxPltCount; i++) {
			xPos = sys->pltInfoVecs.pltLocX[i];
			yPos = sys->pltInfoVecs.pltLocY[i];
			zPos = sys->pltInfoVecs.pltLocZ[i];

			ofs << std::setprecision(5) <<std::fixed<< xPos << " " << yPos << " " << zPos << " " << '\n'<< std::fixed;
		}

		//std::cout<<'here1'<<std::flush;
		
		unsigned numCells = 1;

		unsigned numNumsInCells = 1 + maxPltCount;

		ofs << "CELLS " << numCells << " " << numNumsInCells << std::endl;
		
		//place edges as cells of type 2. 
		ofs<< maxPltCount;
		for (unsigned point = 0; point < maxPltCount; point++ ){
			ofs<< " " << point;
		}
		ofs<<" "<< std::endl;

		ofs << "CELL_TYPES " << numCells << std::endl;  
		//set edges and last set scattered points
				
		ofs << 2 << std::endl;//scatter points for capsid
		
	}
	
	//now print platelets with attatchments
	if ((sys)) {
		unsigned digits = ceil(log10(iteration + 1));
		std::string format = ".vtk";
		std::string Number;
		std::string initial = "AnimationTest/PlateletConn";
		std::ofstream ofs;
		if (digits == 1 || digits == 0) {
			Number = "0000" + std::to_string(iteration);
		}
		else if (digits == 2) {
			Number = "000" + std::to_string(iteration);
		}
		else if (digits == 3) {
			Number = "00" + std::to_string(iteration);
		}
		else if (digits == 4) {
			Number = "0" + std::to_string(iteration);
		}

		std::string Filename = initial + Number + format;

		ofs.open(Filename.c_str());
		
	
		unsigned maxPltCount = sys->generalParams.maxPltCount;

		unsigned num_connections = sys->pltInfoVecs.numConnections;
		
		double xPos;
		double yPos;
		double zPos;
		
		ofs << "# vtk DataFile Version 3.0" << std::endl;
		ofs << "Point representing Sub_cellular elem model" << std::endl;
		ofs << "ASCII" << std::endl << std::endl;
		ofs << "DATASET UNSTRUCTURED_GRID" << std::endl;
		
		 
		ofs << "POINTS " << maxPltCount + num_connections << " float" << std::endl;
		for (unsigned i = 0; i< maxPltCount; i++) {
			xPos = sys->pltInfoVecs.pltLocX[i];
			yPos = sys->pltInfoVecs.pltLocY[i];
			zPos = sys->pltInfoVecs.pltLocZ[i];

			ofs << std::setprecision(5) <<std::fixed<< xPos << " " << yPos << " " << zPos << " " << '\n'<< std::fixed;
		}

		//set location for nodes that plt is connected to
		//ie  
		for (unsigned i = 0; i < num_connections; i++ ) {
			unsigned node_id = sys->pltInfoVecs.nodeImagingConnection[i];

			xPos = sys->nodeInfoVecs.nodeLocX[node_id];
			yPos = sys->nodeInfoVecs.nodeLocY[node_id];
			zPos = sys->nodeInfoVecs.nodeLocZ[node_id];
			
			ofs << std::setprecision(5) <<std::fixed<< xPos << " " << yPos << " " << zPos << " " << '\n'<< std::fixed;
		
		}


		//std::cout<<'here1'<<std::flush;
		
		unsigned numCells = 1;
		numCells += num_connections;//add conections cells for edges

		unsigned numNumsInCells = 1 + maxPltCount;
		numNumsInCells += 3 * num_connections;//3 numbers per edge

		ofs << "CELLS " << numCells << " " << numNumsInCells << std::endl;
		
		//place edges as cells of type 2. 
		ofs<< maxPltCount;
		for (unsigned point = 0; point < maxPltCount; point++ ){
			ofs<< " " << point;
		}
		ofs<<" "<< std::endl;

		
		for (unsigned edge = 0; edge < num_connections; edge++ ){

			//because nodes are placed after platelets, their vtk file id is incremented. 
			//notice that this represents the vtk id, not the id within the c++ program
			unsigned node_id_vtk = maxPltCount + edge;
			unsigned node_id = sys->pltInfoVecs.nodeImagingConnection[edge];
			
			unsigned plt_id = sys->pltInfoVecs.pltImagingConnection[edge];
				
			ofs <<2<< " "<< node_id_vtk << " "<< plt_id <<std::endl;

			//TEMP DELETE AFTER USE
			double locX = sys->nodeInfoVecs.nodeLocX[node_id];
			double locY = sys->nodeInfoVecs.nodeLocY[node_id];
			double locZ = sys->nodeInfoVecs.nodeLocZ[node_id];
			double pltX = sys->pltInfoVecs.pltLocX[plt_id];
			double pltY = sys->pltInfoVecs.pltLocY[plt_id];
			double pltZ = sys->pltInfoVecs.pltLocZ[plt_id];
			double distX = locX - pltX;
			double distY = locY - pltY;
			double distZ = locZ - pltZ;
			 
			double dist = sqrt(distX*distX + distY*distY + distZ * distZ);
			std::cout<<"dist: "<< dist<<std::endl;


		}
		ofs << "CELL_TYPES " << numCells << std::endl;  
		//set edges and last set scattered points
				
		ofs << 2 << std::endl;//scatter points for capsid
		
	//	std::cout<<'here3'<<std::flush;
		for (unsigned edge = 0; edge< num_connections; edge++ ){
			ofs<< 3 <<std::endl;
		}
		ofs.close();
	}

};
