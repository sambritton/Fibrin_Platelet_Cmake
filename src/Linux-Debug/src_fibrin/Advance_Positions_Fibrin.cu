#include "functor_advance_pos.h"

#include "SystemStructures.h"
#include "System.h"
#include "Advance_Positions_Fibrin.h"


void Advance_Positions_Fibrin(
	NodeInfoVecs& nodeInfoVecs,
	GeneralParams& generalParams,
	RandVecs& randVecs) {


		//At this point, the previous node location is the same as the current node,
		//we can therefore use previous node locations to update nodeLoc.
		unsigned _seed = rand();
    	thrust::counting_iterator<unsigned> index_sequence_begin(_seed);

    	thrust::transform(thrust::device, index_sequence_begin, index_sequence_begin + (generalParams.maxNodeCount),
        	randVecs.gaussianData.begin(), psrunifgen(-1.0, 1.0));

		thrust::counting_iterator<unsigned> nodeIndexBegin(0);

		thrust::transform(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeIndexBegin,
					nodeInfoVecs.nodeLocX.begin(),
					nodeInfoVecs.nodeLocY.begin(),
					nodeInfoVecs.nodeLocZ.begin())),
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeIndexBegin,
					nodeInfoVecs.nodeLocX.begin(),
					nodeInfoVecs.nodeLocY.begin(),
					nodeInfoVecs.nodeLocZ.begin())) + generalParams.maxNodeCount,
			//second vector begin
			thrust::make_zip_iterator(
				thrust::make_tuple(
					randVecs.gaussianData.begin(),
					nodeInfoVecs.nodeForceX.begin(),
					nodeInfoVecs.nodeForceY.begin(),
					nodeInfoVecs.nodeForceZ.begin())),
			//save result in third vector to test values
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.nodeLocX.begin(),
					nodeInfoVecs.nodeLocY.begin(),
					nodeInfoVecs.nodeLocZ.begin(),
					nodeInfoVecs.nodeVelocity.begin())),
			functor_advance_pos(generalParams.dtTemp,
				generalParams.viscousDamp_Fibrin,
				generalParams.temperature,
				generalParams.kB,
				generalParams.nodeMass,
				generalParams.maxNodeCount,
				thrust::raw_pointer_cast(nodeInfoVecs.isNodeFixed.data())));

}
